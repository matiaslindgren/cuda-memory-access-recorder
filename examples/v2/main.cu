#include "hip/hip_runtime.h"
/*
 * Pattern analysis applied to example from http://ppc.cs.aalto.fi/ch4/v2/
 */
#include <algorithm>
#include <fstream>
#include <iostream>
#include <iterator>
#include <random>
#include <vector>

#include <hip/hip_runtime.h>

#include "pattern_recorder.cuh"


inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)


const char* patterns_out_path = "access-patterns-v2.json";


template <class KernelData>
__global__
void mykernel(float* r, KernelData d, int n, int nn) {
	d.enter_kernel();
	int ia = threadIdx.x;
	int ja = threadIdx.y;
	int ic = blockIdx.x;
	int jc = blockIdx.y;

	// pr::PatternRecorder and pr::AccessCounter do not currently support pointer arithmetic
	/* const float* t = d + nn * nn; */

	float v[8][8];
	for (int ib = 0; ib < 8; ++ib) {
		for (int jb = 0; jb < 8; ++jb) {
			v[ib][jb] = HUGE_VALF;
		}
	}
	for (int k = 0; k < n; ++k) {
		float x[8];
		float y[8];
		for (int ib = 0; ib < 8; ++ib) {
			int i = ic * 64 + ib * 8 + ia;
			/* x[ib] = t[nn*k + i]; */
			x[ib] = d[nn * nn + nn*k + i];
		}
		for (int jb = 0; jb < 8; ++jb) {
			int j = jc * 64 + jb * 8 + ja;
			y[jb] = d[nn*k + j];
		}
		for (int ib = 0; ib < 8; ++ib) {
			for (int jb = 0; jb < 8; ++jb) {
				v[ib][jb] = min(v[ib][jb], x[ib] + y[jb]);
			}
		}
	}
	for (int ib = 0; ib < 8; ++ib) {
		for (int jb = 0; jb < 8; ++jb) {
			int i = ic * 64 + ib * 8 + ia;
			int j = jc * 64 + jb * 8 + ja;
			if (i < n && j < n) {
				r[n*i + j] = v[ib][jb];
			}
		}
	}
}


__global__ void myppkernel(const float* r, float* d, int n, int nn) {
	int ja = threadIdx.x;
	int i = blockIdx.y;

	float* t = d + nn * nn;

	for (int jb = 0; jb < nn; jb += 64) {
		int j = jb + ja;
		float v = (i < n && j < n) ? r[n*i + j] : HUGE_VALF;
		d[nn*i + j] = v;
		t[nn*j + i] = v;
	}
}


inline int static divup(int a, int b) {
	return (a + b - 1) / b;
}

inline int static roundup(int a, int b) {
	return divup(a, b) * b;
}


void step(float* r, const float* d, int n) {
	int nn = roundup(n, 64);

	// Allocate memory & copy data to GPU
	float* dGPU = NULL;
	CHECK(hipMalloc((void**)&dGPU, 2 * nn * nn * sizeof(float)));
	float* rGPU = NULL;
	CHECK(hipMalloc((void**)&rGPU, n * n * sizeof(float)));
	CHECK(hipMemcpy(rGPU, d, n * n * sizeof(float), hipMemcpyHostToDevice));

	// Run normalization kernel
	{
		dim3 dimBlock(64, 1);
		dim3 dimGrid(1, nn);
		myppkernel<<<dimGrid, dimBlock>>>(rGPU, dGPU, n, nn);
		CHECK(hipGetLastError());
	}

	// Run computation kernel twice to compute access patterns
	{
		dim3 dimBlock(8, 8);
		dim3 dimGrid(nn / 64, nn / 64);
		pr::AccessCounter<float> counter(dGPU, dimGrid);
		mykernel<pr::AccessCounter<float> ><<<dimGrid, dimBlock>>>(rGPU, counter, n, nn);
		CHECK(hipDeviceSynchronize());
		counter.dump_access_statistics(std::cout);
		pr::PatternRecorder<float> recorder(dGPU, dimGrid, counter.get_max_access_count());
		mykernel<pr::PatternRecorder<float> ><<<dimGrid, dimBlock>>>(rGPU, recorder, n, nn);
		CHECK(hipDeviceSynchronize());
		std::ofstream outf(patterns_out_path);
		recorder.dump_json_results(outf, nn, nn);
	}

	CHECK(hipGetLastError());

	// Copy data back to CPU & release memory
	CHECK(hipMemcpy(r, rGPU, n * n * sizeof(float), hipMemcpyDeviceToHost));
	CHECK(hipFree(dGPU));
	CHECK(hipFree(rGPU));
}


__host__
float next_float() {
	static std::random_device rd;
	static std::default_random_engine e(rd());
	static std::uniform_real_distribution<float> floats(0.0, 1.0);
	return floats(e);
}


__host__
int main() {
	// Generate data
	int n = 64;
	std::vector<float> matrix(n * n);
	std::generate(matrix.begin(), matrix.end(), next_float);
	std::vector<float> result(n * n);
	// Compute stuff
	step(result.data(), matrix.data(), n);
	// Write dummy output
	std::ofstream outf("/dev/null");
	std::copy(result.begin(), result.end(), std::ostream_iterator<float>(outf, " "));
}
